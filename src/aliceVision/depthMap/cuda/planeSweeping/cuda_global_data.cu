#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#include "aliceVision/depthMap/cuda/planeSweeping/cuda_global_data.cuh"

#include "aliceVision/depthMap/cuda/deviceCommon/device_color.cuh"

#include <iostream>

// Macro for checking cuda errors
#define CHECK_CUDA_ERROR()                                                    \
    if(hipError_t err = hipGetLastError())                                  \
                                                                              \
{                                                                             \
        fprintf(stderr, "\n\nCUDAError: %s\n", hipGetErrorString(err));      \
        fprintf(stderr, "  file:       %s\n", __FILE__);                      \
        fprintf(stderr, "  function:   %s\n", __FUNCTION__);                  \
        fprintf(stderr, "  line:       %d\n\n", __LINE__);                    \
                                                                              \
}


namespace aliceVision {
namespace depthMap {

/*
 * We keep data in this array that is frequently allocated and freed, as well
 * as recomputed in the original code without a decent need.
 *
 * The code is not capable of dealing with multiple GPUs yet (on multiple GPUs,
 * multiple allocations are probably required).
 */
GlobalData global_data;

// texture<float, hipTextureType1D, hipReadModeElementType> gaussianTex;

void GaussianArray::create( float delta, int radius )
{
    std::cerr << "Computing Gaussian table for radius " << radius << " and delta " << delta << std::endl;

    int size = 2 * radius + 1;

    float* d_gaussian;
    hipMalloc((void**)&d_gaussian, (2 * radius + 1) * sizeof(float));
    CHECK_CUDA_ERROR();

    // generate gaussian array
    generateGaussian_kernel<<<1, size>>>(d_gaussian, delta, radius);
    hipDeviceSynchronize();

    // create cuda array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipMallocArray(&arr, &channelDesc, size, 1);
    CHECK_CUDA_ERROR();
    hipMemcpyToArray(arr, 0, 0, d_gaussian, size * sizeof(float), hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR();
    hipFree(d_gaussian);
    CHECK_CUDA_ERROR();

    hipResourceDesc res_desc;
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = arr;

    hipTextureDesc      tex_desc;
    memset(&tex_desc, 0, sizeof(hipTextureDesc));
    tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    tex_desc.addressMode[0]   = hipAddressModeClamp;
    tex_desc.addressMode[1]   = hipAddressModeClamp;
    tex_desc.addressMode[2]   = hipAddressModeClamp;
    tex_desc.readMode         = hipReadModeElementType; // read as float
    tex_desc.filterMode       = hipFilterModePoint; // apparently default for references
    // tex_desc.filterMode       = hipFilterModeLinear; // no interpolation

    hipCreateTextureObject( &tex, &res_desc, &tex_desc, 0 );
    CHECK_CUDA_ERROR();
}

GlobalData::~GlobalData( )
{
    auto end = _gaussian_arr_table.end();
    for( auto it=_gaussian_arr_table.begin(); it!=end;it++ )
    {
        // cudaDestroyTexture( it->second->tex );
        hipFreeArray( it->second->arr );
    }
}

GaussianArray* GlobalData::getGaussianArray( float delta, int radius )
{
    auto it = _gaussian_arr_table.find( GaussianArrayIndex(radius,delta) );
    if( it != _gaussian_arr_table.end() )
    {
        return it->second;
    }

    GaussianArray* a = new GaussianArray;
    a->create( delta, radius );

    _gaussian_arr_table.insert( std::pair<GaussianArrayIndex,GaussianArray*>( GaussianArrayIndex(radius,delta), a ) );

    return a;
}

void GlobalData::allocScaledPictureArrays( int scales, int ncams, int width, int height )
{
    _scaled_picture_scales = scales;

    _scaled_picture_array.resize( scales * ncams );
    _scaled_picture_tex  .resize( scales * ncams );

    hipResourceDesc res_desc;
    res_desc.resType = hipResourceTypeArray;

    hipTextureDesc      tex_desc;
    memset(&tex_desc, 0, sizeof(hipTextureDesc));
    tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    tex_desc.addressMode[0]   = hipAddressModeClamp;
    tex_desc.addressMode[1]   = hipAddressModeClamp;
    tex_desc.addressMode[2]   = hipAddressModeClamp;
    tex_desc.readMode         = hipReadModeNormalizedFloat;
    tex_desc.filterMode       = hipFilterModeLinear;

    for( int c=0; c<ncams; c++ )
    {
        for( int s=0; s<scales; s++ )
        {
            int w = width / (s + 1);
            int h = height / (s + 1);
            _scaled_picture_array[ c * scales + s ] = new CudaArray<uchar4, 2>( CudaSize<2>( w, h ) );

            res_desc.res.array.array = _scaled_picture_array[ c * scales + s ]->getArray();

            hipCreateTextureObject( &_scaled_picture_tex[ c * scales + s ],
                                     &res_desc,
                                     &tex_desc,
                                     0 );
        }
    }

}

void GlobalData::freeScaledPictureArrays( )
{
    _scaled_picture_scales = 0;

    for( CudaArray<uchar4,2>* ptr : _scaled_picture_array )
    {
        delete ptr;
    }

    _scaled_picture_array.clear();

    for( hipTextureObject_t& obj : _scaled_picture_tex )
    {
        hipDestroyTextureObject( obj );
    }

    _scaled_picture_tex.clear();
}

CudaArray<uchar4,2>* GlobalData::getScaledPictureArrayPtr( int scale, int cam )
{
    return _scaled_picture_array[ cam * _scaled_picture_scales + scale ];
}

CudaArray<uchar4,2>& GlobalData::getScaledPictureArray( int scale, int cam )
{
    return *_scaled_picture_array[ cam * _scaled_picture_scales + scale ];
}

hipTextureObject_t GlobalData::getScaledPictureTex( int scale, int cam )
{
    return _scaled_picture_tex[ cam * _scaled_picture_scales + scale ];
}

void GlobalData::allocPyramidArrays( int levels, int w, int h )
{
    _pyramid_levels = levels;

    _pyramid_array.resize( levels );
    _pyramid_tex  .resize( levels );

    hipTextureDesc      tex_desc;
    memset(&tex_desc, 0, sizeof(hipTextureDesc));
    tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    tex_desc.addressMode[0]   = hipAddressModeClamp;
    tex_desc.addressMode[1]   = hipAddressModeClamp;
    tex_desc.addressMode[2]   = hipAddressModeClamp;
    tex_desc.readMode         = hipReadModeNormalizedFloat;
    tex_desc.filterMode       = hipFilterModeLinear;

    for( int lvl=0; lvl<levels; lvl++ )
    {
        _pyramid_array[ lvl ] = new CudaDeviceMemoryPitched<uchar4, 2>( CudaSize<2>( w, h ) );

        hipResourceDesc res_desc;
        res_desc.resType = hipResourceTypePitch2D;
        res_desc.res.pitch2D.desc         = hipCreateChannelDesc<uchar4>();
        res_desc.res.pitch2D.devPtr       = _pyramid_array[ lvl ]->getBuffer();
        res_desc.res.pitch2D.width        = _pyramid_array[ lvl ]->getSize()[0];
        res_desc.res.pitch2D.height       = _pyramid_array[ lvl ]->getSize()[1];
        res_desc.res.pitch2D.pitchInBytes = _pyramid_array[ lvl ]->getPitch();

        hipCreateTextureObject( &_pyramid_tex[ lvl ],
                                 &res_desc,
                                 &tex_desc,
                                 0 );
        w /= 2;
        h /= 2;
    }
}

void GlobalData::freePyramidArrays( )
{
    _pyramid_levels = 0;

    for( CudaDeviceMemoryPitched<uchar4,2>* ptr : _pyramid_array )
    {
        delete ptr;
    }

    _pyramid_array.clear();

    for( hipTextureObject_t& obj : _pyramid_tex )
    {
        hipDestroyTextureObject( obj );
    }

    _pyramid_tex.clear();
}

CudaDeviceMemoryPitched<uchar4,2>& GlobalData::getPyramidArray( int level )
{
    return *_pyramid_array[ level ];
}

hipTextureObject_t GlobalData::getPyramidTex( int level )
{
    return _pyramid_tex[ level ];
}

PitchedMem_LinearTexture<uchar4>* GlobalData::getPitchedMemUchar4_LinearTexture( int width, int height )
{
    auto it = _pitched_mem_uchar4_linear_tex_cache.find( PitchedMem_Tex_Index( width, height ) );
    if( it == _pitched_mem_uchar4_linear_tex_cache.end() )
    {
        std::cerr << "Allocate pitched mem uchar4 with linear tex " << width << "X" << height << std::endl;
        PitchedMem_LinearTexture<uchar4>* ptr = new PitchedMem_LinearTexture<uchar4>( width, height );
        return ptr;
    }
    else
    {
        std::cerr << "Getting pitched mem uchar4 with linear tex " << width << "X" << height << std::endl;
        PitchedMem_LinearTexture<uchar4>* ptr = it->second;
        _pitched_mem_uchar4_linear_tex_cache.erase( it );
        return ptr;
    }
}

void GlobalData::putPitchedMemUchar4_LinearTexture( PitchedMem_LinearTexture<uchar4>* ptr )
{
    int width  = ptr->mem->getSize()[0];
    int height = ptr->mem->getSize()[1];
    std::cerr << "Putting pitched mem uchar4 with linear tex " << width << "X" << height << std::endl;
    PitchedMem_Tex_Index idx( width, height );
    _pitched_mem_uchar4_linear_tex_cache.insert(
        std::pair<PitchedMem_Tex_Index,PitchedMem_LinearTexture<uchar4>*>(
            idx, ptr ) );
}

}; // namespace depthMap
}; // namespace aliceVision

